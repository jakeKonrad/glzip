#include "hip/hip_runtime.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>

typedef unsigned char uchar;

typedef unsigned int uint;

namespace cg = cooperative_groups;

template <typename OP>
__global__ void adj_map_kernel(
    size_t* d_indptr,
    uchar* d_indices,
    uint v,
    OP op
) {
    // designate this group
    auto tb = cg::this_thread_block();

    __shared__ uint prev_edge;

    __shared__ size_t start; 
    start = d_indptr[v];
    __shared__ size_t end;
   end = d_indptr[v + 1];

    tb.sync();

    if ((end - start) > 0) { 

        // choose a leader to compute the first edge
        if (tb.thread_rank() == 0) {
            // https://github.com/jshun/ligra/blob/master/ligra/byteRLE.h#L64-L83
            uchar const fb = d_indices[start];
            start++;

            uint edgeRead = fb & 0x3f;
	    if (fb & 0x80) {
                int shiftAmount = 6;
		while (true) {
                    uchar b = d_indices[start];
		    edgeRead |= (b & 0x7f) << shiftAmount;
                    start++;
                    if (b & 0x80) {
                        shiftAmount += 7;
                    } else {
                        break;
                    }
                }
            }

            prev_edge = (fb & 0x40) ? v - edgeRead : v + edgeRead;
            op(prev_edge);
        }

        tb.sync();
		    
	__shared__ uint num_bytes;
        __shared__ uint run_length;
	__shared__ uint smem_buffer[64];

	while (start < end) {
	    // parse header
            if (tb.thread_rank() == 0) {
                uchar header = d_indices[start];
	        start++;
	        num_bytes = 1 + (header & 0x3);
                run_length = 1 + (header >> 2);
            }
            tb.sync();

            // compute diffs
            uint diff = 0; 
            for (int i = 0; i < num_bytes; i++) {
                diff = diff << 8;
                diff += (uint) d_indices[start + (threadIdx.x * num_bytes) + i];
            }
            smem_buffer[threadIdx.x] = prev_edge + diff;
            tb.sync();

            auto tile = cg::tiled_partition<8>(tb);

            uint u = cg::inclusive_scan(tile, smem_buffer);

            op(u);

            if (tb.thread_rank() == (run_length - 1)) {
                start += (num_bytes * run_length);
                prev_edge = smem_buffer[threadIdx.x];
            }
            tb.sync();
	}
    }
}

